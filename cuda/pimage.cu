#include "hip/hip_runtime.h"

#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include <>

#include <hip/hip_runtime_api.h>

#define CCE checkCudaErrors

#include <iostream>
#include <stdlib.h>
#include <stdio.h>
#include <math.h>
#include <float.h>
#include <assert.h>

#include "pimage_local.h"

#include "../pimage.h"

using namespace std;

const int gl_win_size = 800;

const double phong_exponent = 1.5;

const double zeye_global = 25;

__device__ int akt_deg_global;


__device__ int gl_max_expo[3];


#include "ppoly.cu"

 __device__ int cudaPoly3::degree()
{
	int i;
	int deg = 0;
	int deg_akt;
	for(i= 0; i < len; ++i) {
		deg_akt = xexpo[i] + yexpo[i] + zexpo[i];
		if (deg_akt > deg) {
			deg = deg_akt;
		}
	}
	return deg;
}

__device__ cudaPoly3 & cudaPoly3::diff(int iv)
{
	int i;
	for(i = 0; i < len; ++i) {
		switch (iv) {
		case 0: coefs[i] *= xexpo[i];
				if (xexpo[i] > 0)--xexpo[i];
				break;
		case 1: coefs[i] *= yexpo[i];
				if (yexpo[i] > 0)--yexpo[i];
				break;
		case 2: coefs[i] *= zexpo[i];
				if (zexpo[i] > 0) --zexpo[i];
				break;
		default: break;
		}
	}

	return *this;

}

__device__ double cudaPoly3::eval(double* sl)
{
	double val = 0;
	int i;

	const int max_expo = 15;

	double xx, yy, zz;
	double xl[max_expo], yl[max_expo], zl[max_expo];

	xx = 1;
	yy = 1;
	zz = 1;

	for(i = 0; i <= gl_max_expo[0]; ++i) {
		xl[i] = xx;
		xx *= sl[0];
	}

	for(i = 0; i <= gl_max_expo[1]; ++i) {
		yl[i] = yy;
		yy *= sl[1];
	}

	for(i = 0; i <= gl_max_expo[2]; ++i) {
		zl[i] = zz;
		zz *= sl[2];
	}

	for(i = 0; i < len; ++i) {
		if (coefs[i] != 0) {
			val += coefs[i] * xl[xexpo[i]] * yl[yexpo[i]] * zl[zexpo[i]];

			//val += coefs[i] * pow(sl[0], xexpo[i]) * pow(sl[1], yexpo[i]) * pow(sl[2], zexpo[i]);
		}
	}
	return val;
}

__device__ void print_cudaPoly3(cudaPoly3 & pol)
{
	for (int i = 0; i < pol.len; ++i) {
		printf("%f * x^%d y^%d z^%d + ", pol.coefs[i], pol.xexpo[i], pol.yexpo[i], pol.zexpo[i]);
	}
	printf("\n deg f = %d \n", pol.degree());
}






inline __device__ int & mref(int* mat, int i, int j)
{
	return mat[2 * i * gl_win_size + j];
}


#define EPS 1e-12
#define M_INF -1e38

__constant__ __device__ cudaPoly3 f3;

static __device__ cudaPoly3 f3x, f3y, f3z;

static __device__ double m_euler[3 * 3];

const int max_deg = 20;

static __device__ int akt_deg;

static __device__ double akt_xbase[max_deg+1] = { 1.0, 1.5, 2.0, 2.5, 3.0, 3.5, 4.0 };

static __device__ double lagrange_basis[max_deg+1][max_deg+1];


__device__ double cuda_skalp_3(double* x, double* y)
{
	return x[0]*y[0] + x[1] * y[1] + x[2] * y[2];
}


__device__ void cuda_skalmult_3(double c, double *x)
{
	x[0] *= c;
	x[1] *= c;
	x[2] *= c;
}

__device__ void cuda_norml_3(double* x)
{
	double len = sqrt(cuda_skalp_3(x,x));
	if (len > 0) {
		cuda_skalmult_3(1/len, x);
	}
}

__device__ void cuda_vecadd_3(double * z, double *x, double *y)
{
	z[0] = x[0] + y[0];
	z[1] = x[1] + y[1];
	z[2] = x[2] + y[2];

}

__device__ void cuda_vecsub_3(double * z, double *x, double *y)
{
	z[0] = x[0] - y[0];
	z[1] = x[1] - y[1];
	z[2] = x[2] - y[2];

}



__device__ double cuda_zscale_factor(double z)
{
	return ((z - zeye_global)/(0 - zeye_global));
}


__device__ void cuda_lb_poly_mult(int deg, double * coefs, double a)
{
	// multiply coefs poly coefs[0] * t^deg + ... with (t + a)
	coefs[deg+1] = 0;
	for(int i = deg + 1; i >=  1; --i) {
		coefs[i] += a * coefs[i-1];
	}
}

__device__ void cuda_lb_poly_scal_mult(int deg, double * coefs, double a)
{
	for(int i = 0; i <= deg; ++i) {
		coefs[i] *= a;
	}
}


__device__ void cuda_lb_gen_lagrange_basis(int deg, double* xbase, double lagr_basis[][max_deg + 1])
{
	// xbase stützstellen, deg+1 Stück

	for(int i = 0; i <= deg; ++i) {
		double aux_poly[max_deg + 1];

		aux_poly[0] = 1;
		double pre_coef = 1;
		int curr_deg = 0;

		for(int j = 0; j <= deg; ++j) {
			if (i == j) {
				continue;
			}
			pre_coef *= (xbase[i] - xbase[j]);
			cuda_lb_poly_mult(curr_deg, aux_poly, -xbase[j]);
			++curr_deg;
		}
		cuda_lb_poly_scal_mult(deg, aux_poly, 1/pre_coef);

		memcpy(lagr_basis[i], aux_poly, sizeof(aux_poly));

	}
}

__device__ void cuda_m_mult_vec_mat(double* vec, double *mat, double * vec_res) {
	vec_res[0] = vec[0] * mat[0] + vec[1] * mat[3] + vec[2] * mat[6];
	vec_res[1] = vec[0] * mat[1] + vec[1] * mat[4] + vec[2] * mat[7];
	vec_res[2] = vec[0] * mat[2] + vec[1] * mat[5] + vec[2] * mat[8];
}

__device__ void cuda_m_mult_mat_vec(double* vec, double *mat, double * vec_res) {
	vec_res[0] = vec[0] * mat[0] + vec[1] * mat[1] + vec[2] * mat[2];
	vec_res[1] = vec[0] * mat[3] + vec[1] * mat[4] + vec[2] * mat[5];
	vec_res[2] = vec[0] * mat[6] + vec[1] * mat[7] + vec[2] * mat[8];
}


__device__ void cuda_m_rot_z(double a, double * mat)
{
	const int ndim = 3;
	for(int i = 0; i < ndim; ++i) {
		for(int j = 0; j < ndim; ++j) {
			mat[3*i+j] = 0;
		}
	}
	mat[0] = cos(a);
	mat[1] = sin(a);
	mat[3] = -sin(a);
	mat[4] = cos(a);
	mat[8] = 1;
}

__device__ void cuda_m_rot_x(double a, double * mat)
{
	const int ndim = 3;
	for(int i = 0; i < ndim; ++i) {
		for(int j = 0; j < ndim; ++j) {
			mat[3*i+j] = 0;
		}
	}
	mat[4] = cos(a);
	mat[5] = sin(a);
	mat[7] = -sin(a);
	mat[8] = cos(a);
	mat[0] = 1;

}

__device__ void cuda_m_mult(double * m1, double * m2, double * m3)
{
	const int ndim = 3;
	for(int i = 0; i < ndim; ++i) {
		for(int j = 0; j < ndim; ++j) {
			double sum = 0;
			for(int k = 0; k < ndim; ++k) {
				sum += m1[i*3 + k] * m2[k*3 + j];
			}
			m3[i*3 + j] = sum;
		}
	}
}

__device__ void cuda_m_print(double* m)
{
	for(int i = 0; i < 3; ++i) {
		for(int j = 0; j < 3; ++j) {
			printf("%f , ", m[i*3+j]);
		}
		printf("\n");
	}
}

__device__ double cuda_eval_f3_mat(cudaPoly3 & f3, double *sl)
{
	double sl1[3];
	cuda_m_mult_mat_vec(sl, m_euler, sl1);

	double f = f3.eval(sl1);

	return f;

}


__device__ int cuda_eval_poly_poly(cudaPoly3 & f3,
		double x, double y, double z, double & f, double & fx, double & fy, double & fz)
{

	// hier muss
	// 1) sl mit m_euler substituiert werden vor dem Einsetzen
	// 2) (fx fy fz) * m_euler multipliziert werden

	double sl[3];
	double sl1[3];
	sl[0] = x;
	sl[1] = y;
	sl[2] = z;

	cuda_m_mult_mat_vec(sl, m_euler, sl1);

	f = f3.eval(sl1);

	sl[0] = f3x.eval(sl1);
	sl[1] = f3y.eval(sl1);
	sl[2] = f3z.eval(sl1);

	cuda_m_mult_vec_mat(sl, m_euler, sl1);

	fx = sl1[0];
	fy = sl1[1];
	fz = sl1[2];

	return 0;
}

#if 0

__device__ int cuda_eval_coefs_poly(cudaPoly3 & f3, double x, double y, double & a0, double & a1, double & a2, double & a3,
		double & a4)
{
	// hier die Lagrangeinterpolation einfügen
	// z wird nacheinander z0, z1, z2,....zn gesetzt mit n = deg f3
	// m_euler wird zur Substitution der x y z benutzt

	double sl[3];
	sl[0] = x;
	sl[1] = y;
	double coefs[max_deg + 1];
	for(int i = 0; i <= max_deg; ++i) {
		coefs[i] = 0;
	}
	for(int i = 0; i <= akt_deg; ++i) {
		double z = akt_xbase[i];
		sl[2] = z;
		double fi = cuda_eval_f3_mat(f3, sl);
		for(int j = 0; j <= akt_deg; ++j) {
			coefs[j] += fi * lagrange_basis[i][j];
		}
	}

	a0 = coefs[4];
	a1 = coefs[3];
	a2 = coefs[2];
	a3 = coefs[1];
	a4 = coefs[0];
	return 0;
}

#endif

__device__ int cuda_eval_coefs_poly(cudaPoly3 & f3, double x, double y, double* coefs_lis)
{
	// hier die Lagrangeinterpolation einfügen
	// z wird nacheinander z0, z1, z2,....zn gesetzt mit n = deg f3
	// m_euler wird zur Substitution der x y z benutzt

	double sl[3];
	sl[0] = x;
	sl[1] = y;
	double coefs[max_deg + 1];
	for(int i = 0; i <= max_deg; ++i) {
		coefs[i] = 0;
	}
	for(int i = 0; i <= akt_deg_global; ++i) {
		double z = akt_xbase[i];
		sl[2] = z;
		double fi = cuda_eval_f3_mat(f3, sl);
		for(int j = 0; j <= akt_deg_global; ++j) {
			coefs[j] += fi * lagrange_basis[i][j];
		}
	}

	memcpy(coefs_lis, coefs, sizeof(coefs));

	return 0;
}


__device__ int cuda_eval_coefs_poly_centr_persp(cudaPoly3 & f3, double x, double y, double* coefs_lis)
{
	// hier die Lagrangeinterpolation einfügen
	// z wird nacheinander z0, z1, z2,....zn gesetzt mit n = deg f3
	// m_euler wird zur Substitution der x y z benutzt

	double sl[3];
	double coefs[max_deg + 1];
	for(int i = 0; i <= max_deg; ++i) {
		coefs[i] = 0;
	}
	for(int i = 0; i <= akt_deg_global; ++i) {
		double z = akt_xbase[i];

		double zscale_factor = cuda_zscale_factor(z);

		sl[0] = x * zscale_factor;
		sl[1] = y * zscale_factor;
		sl[2] = z;
		double fi = cuda_eval_f3_mat(f3, sl);
		for(int j = 0; j <= akt_deg_global; ++j) {
			coefs[j] += fi * lagrange_basis[i][j];
		}
	}

	memcpy(coefs_lis, coefs, sizeof(coefs));

	return 0;
}





__global__ void cuda_rotate_mat(double phi, double theta, double psi)
{
	const int ndim = 3;
	const int nsize = ndim * ndim;

	double m_z_phi[nsize];
	double m_x_theta[nsize];
	double m_z_psi[nsize];

	double m_aux[nsize];

	cuda_m_rot_z(phi, m_z_phi);
	cuda_m_rot_x(theta, m_x_theta);
	cuda_m_rot_z(psi, m_z_psi);

	cuda_m_mult(m_x_theta, m_z_phi, m_aux);
	cuda_m_mult(m_z_psi, m_aux, m_euler);


}

__global__ void cuda_init_f3_diff()
{
	f3x = f3;
	f3x.diff(0);
	f3y = f3;
	f3y.diff(1);
	f3z = f3;
	f3z.diff(2);

/*
	printf("f3x = ");
	print_cudaPoly3(f3x);
	printf("f3y = ");
	print_cudaPoly3(f3y);
	printf("f3z = ");
	print_cudaPoly3(f3z);
*/

	akt_deg = f3.degree();

	cuda_lb_gen_lagrange_basis(akt_deg, akt_xbase, lagrange_basis);

}




const double clip_radius = 8; //20;

__device__ inline bool cuda_in_clip_radius(double x, double y, double z)
{
	return x * x + y * y + z * z <= clip_radius * clip_radius;
}

__device__ void cuda_normalize_poly_coefs(double* poly_coefs, int akt_deg, int & deg_new)
{

	// in poly_coefs poly_coefs[0] is coefficient in term of highest degree

	int deg;
	int i_tst;

	for(i_tst = 0; i_tst < akt_deg; ++i_tst) {
		if (fabs(poly_coefs[i_tst]) > 1e-8)
			break;
	}

	deg = akt_deg - i_tst;

	deg_new = deg;

	if (deg < akt_deg) {
		for(int i = 0; i < max_deg - (akt_deg - deg); ++i ) {
			poly_coefs[i] = poly_coefs[i + akt_deg - deg];
		}
	}

}


__device__ int cuda_get_z_intersect_poly(double x, double y, double *z, double *n_xyz, bool & disc_zero)
{

#if 0
	double a0 = 0;
	double a1 = 0;
	double a2 = 0;
	double a3 = 0;
	double a4 = 0;
	double poly_coefs[11];

	double z_erg;
	double z_erg_new = M_INF;
	double z_erg_list[11];

	cuda_eval_coefs_poly(f3, x, y, a0, a1, a2, a3, a4);

	poly_coefs[0] = a4;
	poly_coefs[1] = a3;
	poly_coefs[2] = a2;
	poly_coefs[3] = a1;
	poly_coefs[4] = a0;

	int deg = (a4 != 0) ? 4 : (a3 != 0) ? 3 : (a2 != 0) ? 2 : 1;
	int num_z_erg;
#endif

	double poly_coefs[max_deg + 1];

	double z_erg;
	double z_erg_new = M_INF;
	double z_erg_list[max_deg + 1];

	//cuda_eval_coefs_poly(f3, x, y, poly_coefs);
	cuda_eval_coefs_poly_centr_persp(f3, x, y, poly_coefs);


	// coefficient of leading monomial is in poly_coefs[0]
	// akt_deg is intended degree

	int deg;
	int num_z_erg;

	cuda_normalize_poly_coefs(poly_coefs, akt_deg_global, deg);


#if 0
	if (deg < 4) {
		for(int i = 0; i < 6; ++i ) {
			poly_coefs[i] = poly_coefs[i + 4 - deg];
		}
	}

	double disc_poly = x_comp_disc(deg, poly_coefs);

#endif

	if (1) {

		x_root_final_list(deg, poly_coefs, 20, z_erg_list, num_z_erg);

		int j;

		j = num_z_erg - 1;


		while (j >= 0) {

			double zsf = cuda_zscale_factor(z_erg_list[j]);

			double x1 = x * zsf;
			double y1 = y * zsf;

			if (cuda_in_clip_radius(x1,y1,z_erg_list[j])) {
				//cout << "num_z_erg = " << num_z_erg << " j = " << j << " z_erg_new = " << z_erg_list[j] << endl;
				z_erg_new = z_erg_list[j];
				break;
			}
			--j;
		}

	} else {
		z_erg_new = M_INF;
	}

	z_erg = z_erg_new;

	double zsf = cuda_zscale_factor(z_erg);

	double x1 = x * zsf;
	double y1 = y * zsf;



	if (! cuda_in_clip_radius(x1, y1, z_erg)) {
		z_erg = M_INF;
	}

	*z = z_erg;

	if (z_erg > M_INF) {
		double f, fx, fy, fz;
		cuda_eval_poly_poly(f3, x1, y1, z_erg, f, fx, fy, fz);
		n_xyz[0] = fx;
		n_xyz[1] = fy;
		n_xyz[2] = fz;
		cuda_norml_3(n_xyz);
		//*n_z = fz/sqrt(fx*fx+fy*fy+fz*fz);
	} else {
		n_xyz[0] = 0;
		n_xyz[1] = 0;
		n_xyz[2] = 0;
	}

	return 0;
}


const int win_size = gl_win_size;

#define RAND_COL ((int)(255.0*((float)rand())/RAND_MAX))

/*
double z_buf[win_size][win_size];
double n_buf[win_size][win_size];
*/
//#define SCALE 10.0
#define SCALE 50.0



__global__ void compute_colmat(double a, double b, int xmax, int ymax,
		double euler_phi, double euler_theta, double euler_psi,
		int *colmat_r_d, int *colmat_g_d, int *colmat_b_d)
{

	int xx = threadIdx.x + blockIdx.x * blockDim.x;
	int yy = threadIdx.y + blockIdx.y * blockDim.y;

	double n;
	double x1, y1;
	double z;
	double n_xyz[3];
	double local_scale;

	int win_offset;
	bool disc_zero;

	double color_red, color_green, color_blue;
	double phong_kernel, spec_coef;
	int col_red, col_green, col_blue;

	int col_z;

	win_offset = xmax/2;


	local_scale = gl_win_size/xmax;

	//y1 = (yy - win_offset)/SCALE * local_scale;
    //x1 = (xx - win_offset)/SCALE * local_scale;

	y1 = (yy - win_offset)/(0.8 * SCALE) * local_scale;
    x1 = (xx - win_offset)/(0.8 * SCALE) * local_scale;



    //printf("x1 = %f, y1 = %f ", x1, y1);

    cuda_get_z_intersect_poly(x1, y1, &z, n_xyz, disc_zero);


/*
    z_buf[x][y] = z;

    if (z > M_INF) {
    	n_buf[x][y] = n_z;
    } else {
    	n_buf[x][y] = 0;
    }
*/



    // n is cos(i) for light incident along z axis from infinity
	n = (z > M_INF) ? n_xyz[2] : 0;

	double Lin[3] = {0,0,1};

	double Lin_p[3];

	memcpy(Lin_p, n_xyz, sizeof(Lin_p));

	cuda_skalmult_3(2 * cuda_skalp_3(Lin, n_xyz), Lin_p);

	double Lout[3];

	cuda_vecsub_3(Lout, Lin_p, Lin);

	double zsf = cuda_zscale_factor(z);

	double x11 = x1 * zsf;
	double y11 = y1 * zsf;

	double Peye[3] = {0, 0, zeye_global};

	double Psurf[3] = {x11, y11, z};

	double Pvec[3];

	cuda_vecsub_3(Pvec, Peye, Psurf);

	cuda_norml_3(Pvec);

	double coss = cuda_skalp_3(Pvec, Lout);

	assert(fabs(coss) <= 1.0);

	if (z > M_INF) {
		color_red = 0.0;
		color_green = 0.0;
		color_blue = 0.0;

		if (n < 0) {
			color_red = -n/2 + 0.1;
			color_green = 0;
			color_blue = 0.0;
		} else if (n >= 0) {
			color_green = n/2 + 0.1;
			color_red = 0;
			color_blue = 0.0;
		}

		//phong_kernel = 2 * n * n - 1;
		//spec_coef = 0.3 * pow(phong_kernel, phong_exponent);

		spec_coef = coss > 0 ? 0.3 * pow(coss, phong_exponent) : 0.0;

		color_red += spec_coef;
		color_green += spec_coef ;
		color_blue += spec_coef;

		col_red = (int) (250 * color_red);
		col_green = (int) (250 * color_green);
		col_blue = (int) (250 * color_blue);

#if 0

#define MAX_COL_Z (1 << 20)
#define NUM_STRIPES 32
#define STRIPE_PART 8

		col_z = (int)((z/15 + 1.0) * MAX_COL_Z);
		col_z %= MAX_COL_Z/NUM_STRIPES;
		if (0 <= col_z  && col_z <= MAX_COL_Z/(NUM_STRIPES * STRIPE_PART)) {
			col_blue = ::max(col_red, col_green);
			col_red = 0;
			col_green = 0;
		};

#endif

		mref(colmat_r_d, xx, yy) = col_red;
		mref(colmat_g_d, xx, yy) = col_green;
		mref(colmat_b_d, xx, yy) = col_blue;
	} else {
		// background color
		mref(colmat_r_d, xx, yy) = 250; //64;
		mref(colmat_g_d, xx, yy) = 250; //32;
		mref(colmat_b_d, xx, yy) = 250; //64;

	}

	__syncthreads();

}

#define THREAD_NUMXY 16

void gpu_compute_colmat(double a, double b, int xmax, int ymax, const cudaPoly3 & f3_h,
		double euler_phi, double euler_theta, double euler_psi,
		int *colmat_r, int *colmat_g, int *colmat_b) {

	int *colmat_r_d;
	int *colmat_g_d;
	int *colmat_b_d;
	
	int akt_deg_global_host = -1;
	
	int gl_max_expo_host[3] = {-1, -1, -1};

	for(int i = 0; i < 40; ++i) {
		int degi = f3_h.xexpo[i] + f3_h.yexpo[i] + f3_h.zexpo[i];
		if (degi > akt_deg_global_host) {
			akt_deg_global_host = degi;
		}
		if (f3_h.xexpo[i] > gl_max_expo_host[0]) {
			gl_max_expo_host[0] = f3_h.xexpo[i];
		}
		if (f3_h.yexpo[i] > gl_max_expo_host[1]) {
			gl_max_expo_host[1] = f3_h.yexpo[i];
		}
		if (f3_h.zexpo[i] > gl_max_expo_host[2]) {
			gl_max_expo_host[2] = f3_h.zexpo[i];
		}
	}

	const int N = 4 * gl_win_size * gl_win_size;

	cout << "N = " << N << endl;

	cout << "xmax = " << xmax << " ymax = " << ymax << endl;

	//print_cudaPoly3(f3_h);

	printf("sizeof(cudaPoly3) = %ld\n ", sizeof(cudaPoly3));

	x_prepare_binom<<<1,1>>>();

	hipDeviceSynchronize();

	//x_print_binom<<<1,1>>>();

	CCE(hipDeviceSetLimit(hipLimitStackSize , 128 * 1024));

	CCE( hipMemcpyToSymbol(HIP_SYMBOL(akt_deg_global), &akt_deg_global_host, sizeof(int),
					0, hipMemcpyHostToDevice ) );

	CCE( hipMemcpyToSymbol(HIP_SYMBOL(gl_max_expo), &gl_max_expo_host[0], 3 * sizeof(int) ) );



	CCE( hipMemcpyToSymbol(HIP_SYMBOL(f3), &f3_h, sizeof(cudaPoly3), 0, hipMemcpyHostToDevice) );

	hipDeviceSynchronize();


    cuda_rotate_mat<<<1,1>>>(euler_phi, euler_theta, euler_psi);

	cuda_init_f3_diff<<<1,1>>>();

	hipDeviceSynchronize();


	CCE( hipMalloc((void**) &colmat_r_d, N * sizeof(int) ) );
	CCE( hipMalloc((void**) &colmat_g_d, N * sizeof(int) ) );
	CCE( hipMalloc((void**) &colmat_b_d, N * sizeof(int) ) );

	dim3 grids(gl_win_size/THREAD_NUMXY, gl_win_size/THREAD_NUMXY);
	dim3 threads(THREAD_NUMXY, THREAD_NUMXY);

	compute_colmat<<<grids, threads>>>(a, b, xmax, ymax, euler_phi, euler_theta, euler_psi,
			colmat_r_d, colmat_g_d, colmat_b_d);

	cout << "gpu computation done." << endl;


	CCE(hipMemcpy(colmat_r, colmat_r_d, N * sizeof(int), hipMemcpyDeviceToHost));
	CCE(hipMemcpy(colmat_g, colmat_g_d, N * sizeof(int), hipMemcpyDeviceToHost));
	CCE(hipMemcpy(colmat_b, colmat_b_d, N * sizeof(int), hipMemcpyDeviceToHost));

	CCE(hipFree(colmat_r_d));
	CCE(hipFree(colmat_g_d));
	CCE(hipFree(colmat_b_d));


}


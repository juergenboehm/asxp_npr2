#include "hip/hip_runtime.h"

#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include <>

#include <hip/hip_runtime_api.h>

#define CCE checkCudaErrors

#include <iostream>
#include <stdlib.h>
#include <stdio.h>
#include <math.h>
#include <float.h>

#include "pimage_local.h"

#include "../pimage.h"

using namespace std;

const int gl_win_size = 800;

const int phong_exponent = 32;

__device__ int akt_deg_global;

#include "ppoly.cu"

__host__ __device__ int cudaPoly3::degree()
{
	int i;
	int deg = 0;
	int deg_akt;
	for(i= 0; i < len; ++i) {
		deg_akt = xexpo[i] + yexpo[i] + zexpo[i];
		if (deg_akt > deg) {
			deg = deg_akt;
		}
	}
	return deg;
}

__host__ __device__ cudaPoly3 & cudaPoly3::diff(int iv)
{
	int i;
	for(i = 0; i < len; ++i) {
		switch (iv) {
		case 0: coefs[i] *= xexpo[i];
				if (xexpo[i] > 0)--xexpo[i];
				break;
		case 1: coefs[i] *= yexpo[i];
				if (yexpo[i] > 0)--yexpo[i];
				break;
		case 2: coefs[i] *= zexpo[i];
				if (zexpo[i] > 0) --zexpo[i];
				break;
		default: break;
		}
	}

	return *this;

}

__host__ __device__ double cudaPoly3::eval(double* sl)
{
	double val = 0;
	int i;
	for(i = 0; i < len; ++i) {
		if (coefs[i] != 0) {
			val += coefs[i] * pow(sl[0], xexpo[i]) * pow(sl[1], yexpo[i]) * pow(sl[2], zexpo[i]);
		}
	}
	return val;
}

__host__ __device__ void print_cudaPoly3(cudaPoly3 & pol)
{
	for (int i = 0; i < pol.len; ++i) {
		printf("%f * x^%d y^%d z^%d + ", pol.coefs[i], pol.xexpo[i], pol.yexpo[i], pol.zexpo[i]);
	}
	printf("\n deg f = %d \n", pol.degree());
}






inline __device__ int & mref(int* mat, int i, int j)
{
	return mat[2 * i * gl_win_size + j];
}


#define EPS 1e-12
#define M_INF -1e38

__constant__ __device__ cudaPoly3 f3;

static __device__ cudaPoly3 f3x, f3y, f3z;

static __device__ double m_euler[3 * 3];

const int max_deg = 20;

static __device__ int akt_deg;

static __device__ double akt_xbase[max_deg+1] = { 1.0, 1.5, 2.0, 2.5, 3.0, 3.5, 4.0 };

static __device__ double lagrange_basis[max_deg+1][max_deg+1];

__device__ void cuda_lb_poly_mult(int deg, double * coefs, double a)
{
	// multiply coefs poly coefs[0] * t^deg + ... with (t + a)
	coefs[deg+1] = 0;
	for(int i = deg + 1; i >=  1; --i) {
		coefs[i] += a * coefs[i-1];
	}
}

__device__ void cuda_lb_poly_scal_mult(int deg, double * coefs, double a)
{
	for(int i = 0; i <= deg; ++i) {
		coefs[i] *= a;
	}
}


__device__ void cuda_lb_gen_lagrange_basis(int deg, double* xbase, double lagr_basis[][max_deg + 1])
{
	// xbase stützstellen, deg+1 Stück

	for(int i = 0; i <= deg; ++i) {
		double aux_poly[max_deg + 1];

		aux_poly[0] = 1;
		double pre_coef = 1;
		int curr_deg = 0;

		for(int j = 0; j <= deg; ++j) {
			if (i == j) {
				continue;
			}
			pre_coef *= (xbase[i] - xbase[j]);
			cuda_lb_poly_mult(curr_deg, aux_poly, -xbase[j]);
			++curr_deg;
		}
		cuda_lb_poly_scal_mult(deg, aux_poly, 1/pre_coef);

		memcpy(lagr_basis[i], aux_poly, sizeof(aux_poly));

	}
}

__device__ void cuda_m_mult_vec_mat(double* vec, double *mat, double * vec_res) {
	vec_res[0] = vec[0] * mat[0] + vec[1] * mat[3] + vec[2] * mat[6];
	vec_res[1] = vec[0] * mat[1] + vec[1] * mat[4] + vec[2] * mat[7];
	vec_res[2] = vec[0] * mat[2] + vec[1] * mat[5] + vec[2] * mat[8];
}

__device__ void cuda_m_mult_mat_vec(double* vec, double *mat, double * vec_res) {
	vec_res[0] = vec[0] * mat[0] + vec[1] * mat[1] + vec[2] * mat[2];
	vec_res[1] = vec[0] * mat[3] + vec[1] * mat[4] + vec[2] * mat[5];
	vec_res[2] = vec[0] * mat[6] + vec[1] * mat[7] + vec[2] * mat[8];
}


__device__ void cuda_m_rot_z(double a, double * mat)
{
	const int ndim = 3;
	for(int i = 0; i < ndim; ++i) {
		for(int j = 0; j < ndim; ++j) {
			mat[3*i+j] = 0;
		}
	}
	mat[0] = cos(a);
	mat[1] = sin(a);
	mat[3] = -sin(a);
	mat[4] = cos(a);
	mat[8] = 1;
}

__device__ void cuda_m_rot_x(double a, double * mat)
{
	const int ndim = 3;
	for(int i = 0; i < ndim; ++i) {
		for(int j = 0; j < ndim; ++j) {
			mat[3*i+j] = 0;
		}
	}
	mat[4] = cos(a);
	mat[5] = sin(a);
	mat[7] = -sin(a);
	mat[8] = cos(a);
	mat[0] = 1;

}

__device__ void cuda_m_mult(double * m1, double * m2, double * m3)
{
	const int ndim = 3;
	for(int i = 0; i < ndim; ++i) {
		for(int j = 0; j < ndim; ++j) {
			double sum = 0;
			for(int k = 0; k < ndim; ++k) {
				sum += m1[i*3 + k] * m2[k*3 + j];
			}
			m3[i*3 + j] = sum;
		}
	}
}

__device__ void cuda_m_print(double* m)
{
	for(int i = 0; i < 3; ++i) {
		for(int j = 0; j < 3; ++j) {
			printf("%f , ", m[i*3+j]);
		}
		printf("\n");
	}
}

__device__ double cuda_eval_f3_mat(cudaPoly3 & f3, double *sl)
{
	double sl1[3];
	cuda_m_mult_mat_vec(sl, m_euler, sl1);

	double f = f3.eval(sl1);

	return f;

}


__device__ int cuda_eval_poly_poly(cudaPoly3 & f3,
		double x, double y, double z, double & f, double & fx, double & fy, double & fz)
{

	// hier muss
	// 1) sl mit m_euler substituiert werden vor dem Einsetzen
	// 2) (fx fy fz) * m_euler multipliziert werden

	double sl[3];
	double sl1[3];
	sl[0] = x;
	sl[1] = y;
	sl[2] = z;

	cuda_m_mult_mat_vec(sl, m_euler, sl1);

	f = f3.eval(sl1);

	sl[0] = f3x.eval(sl1);
	sl[1] = f3y.eval(sl1);
	sl[2] = f3z.eval(sl1);

	cuda_m_mult_vec_mat(sl, m_euler, sl1);

	fx = sl1[0];
	fy = sl1[1];
	fz = sl1[2];

	return 0;
}

#if 0

__device__ int cuda_eval_coefs_poly(cudaPoly3 & f3, double x, double y, double & a0, double & a1, double & a2, double & a3,
		double & a4)
{
	// hier die Lagrangeinterpolation einfügen
	// z wird nacheinander z0, z1, z2,....zn gesetzt mit n = deg f3
	// m_euler wird zur Substitution der x y z benutzt

	double sl[3];
	sl[0] = x;
	sl[1] = y;
	double coefs[max_deg + 1];
	for(int i = 0; i <= max_deg; ++i) {
		coefs[i] = 0;
	}
	for(int i = 0; i <= akt_deg; ++i) {
		double z = akt_xbase[i];
		sl[2] = z;
		double fi = cuda_eval_f3_mat(f3, sl);
		for(int j = 0; j <= akt_deg; ++j) {
			coefs[j] += fi * lagrange_basis[i][j];
		}
	}

	a0 = coefs[4];
	a1 = coefs[3];
	a2 = coefs[2];
	a3 = coefs[1];
	a4 = coefs[0];
	return 0;
}

#endif

__device__ int cuda_eval_coefs_poly(cudaPoly3 & f3, double x, double y, double* coefs_lis)
{
	// hier die Lagrangeinterpolation einfügen
	// z wird nacheinander z0, z1, z2,....zn gesetzt mit n = deg f3
	// m_euler wird zur Substitution der x y z benutzt

	double sl[3];
	sl[0] = x;
	sl[1] = y;
	double coefs[max_deg + 1];
	for(int i = 0; i <= max_deg; ++i) {
		coefs[i] = 0;
	}
	for(int i = 0; i <= akt_deg_global; ++i) {
		double z = akt_xbase[i];
		sl[2] = z;
		double fi = cuda_eval_f3_mat(f3, sl);
		for(int j = 0; j <= akt_deg_global; ++j) {
			coefs[j] += fi * lagrange_basis[i][j];
		}
	}

	memcpy(coefs_lis, coefs, sizeof(coefs));

	return 0;
}


__global__ void cuda_rotate_mat(double phi, double theta, double psi)
{
	const int ndim = 3;
	const int nsize = ndim * ndim;

	double m_z_phi[nsize];
	double m_x_theta[nsize];
	double m_z_psi[nsize];

	double m_aux[nsize];

	cuda_m_rot_z(phi, m_z_phi);
	cuda_m_rot_x(theta, m_x_theta);
	cuda_m_rot_z(psi, m_z_psi);

	cuda_m_mult(m_x_theta, m_z_phi, m_aux);
	cuda_m_mult(m_z_psi, m_aux, m_euler);


}

__global__ void cuda_init_f3_diff()
{
	f3x = f3;
	f3x.diff(0);
	f3y = f3;
	f3y.diff(1);
	f3z = f3;
	f3z.diff(2);

/*
	printf("f3x = ");
	print_cudaPoly3(f3x);
	printf("f3y = ");
	print_cudaPoly3(f3y);
	printf("f3z = ");
	print_cudaPoly3(f3z);
*/

	akt_deg = f3.degree();

	cuda_lb_gen_lagrange_basis(akt_deg, akt_xbase, lagrange_basis);

}




const double clip_radius = 8; //20;

__device__ inline bool cuda_in_clip_radius(double x, double y, double z)
{
	return x * x + y * y + z * z <= clip_radius * clip_radius;
}

__device__ void cuda_normalize_poly_coefs(double* poly_coefs, int akt_deg, int & deg_new)
{

	// in poly_coefs poly_coefs[0] is coefficient in term of highest degree

	int deg;
	int i_tst;

	for(i_tst = 0; i_tst < akt_deg; ++i_tst) {
		if (fabs(poly_coefs[i_tst]) > 1e-8)
			break;
	}

	deg = akt_deg - i_tst;

	deg_new = deg;

	if (deg < akt_deg) {
		for(int i = 0; i < max_deg - (akt_deg - deg); ++i ) {
			poly_coefs[i] = poly_coefs[i + akt_deg - deg];
		}
	}

}


__device__ int cuda_get_z_intersect_poly(double x, double y, double *z, double *n_z, bool & disc_zero)
{

#if 0
	double a0 = 0;
	double a1 = 0;
	double a2 = 0;
	double a3 = 0;
	double a4 = 0;
	double poly_coefs[11];

	double z_erg;
	double z_erg_new = M_INF;
	double z_erg_list[11];

	cuda_eval_coefs_poly(f3, x, y, a0, a1, a2, a3, a4);

	poly_coefs[0] = a4;
	poly_coefs[1] = a3;
	poly_coefs[2] = a2;
	poly_coefs[3] = a1;
	poly_coefs[4] = a0;

	int deg = (a4 != 0) ? 4 : (a3 != 0) ? 3 : (a2 != 0) ? 2 : 1;
	int num_z_erg;
#endif

	double poly_coefs[max_deg + 1];

	double z_erg;
	double z_erg_new = M_INF;
	double z_erg_list[max_deg + 1];

	cuda_eval_coefs_poly(f3, x, y, poly_coefs);

	// coefficient of leading monomial is in poly_coefs[0]
	// akt_deg is intended degree

	int deg;
	int num_z_erg;

	cuda_normalize_poly_coefs(poly_coefs, akt_deg_global, deg);


#if 0
	if (deg < 4) {
		for(int i = 0; i < 6; ++i ) {
			poly_coefs[i] = poly_coefs[i + 4 - deg];
		}
	}

	double disc_poly = x_comp_disc(deg, poly_coefs);

#endif

	if (1) {

		x_root_final_list(deg, poly_coefs, 20, z_erg_list, num_z_erg);

		int j;

		j = num_z_erg - 1;
		while (j >= 0) {
			if (cuda_in_clip_radius(x,y,z_erg_list[j])) {
				//cout << "num_z_erg = " << num_z_erg << " j = " << j << " z_erg_new = " << z_erg_list[j] << endl;
				z_erg_new = z_erg_list[j];
				break;
			}
			--j;
		}

	} else {
		z_erg_new = M_INF;
	}

	z_erg = z_erg_new;


	if (! cuda_in_clip_radius(x, y, z_erg)) {
		z_erg = M_INF;
	}

	*z = z_erg;

	if (z_erg > M_INF) {
		double f, fx, fy, fz;
		cuda_eval_poly_poly(f3, x, y, z_erg, f, fx, fy, fz);
		*n_z = fz/sqrt(fx*fx+fy*fy+fz*fz);
	} else {
		*n_z = 0;
	}

	return 0;
}


const int win_size = gl_win_size;

#define RAND_COL ((int)(255.0*((float)rand())/RAND_MAX))

/*
double z_buf[win_size][win_size];
double n_buf[win_size][win_size];
*/
//#define SCALE 10.0
#define SCALE 50.0



__global__ void compute_colmat(double a, double b, int xmax, int ymax,
		double euler_phi, double euler_theta, double euler_psi,
		int *colmat_r_d, int *colmat_g_d, int *colmat_b_d)
{

	int xx = threadIdx.x + blockIdx.x * blockDim.x;
	int yy = threadIdx.y + blockIdx.y * blockDim.y;

	double n;
	double x1, y1;
	double z, n_z;
	double local_scale;

	int win_offset;
	bool disc_zero;

	double color_red, color_green, color_blue;
	double phong_kernel, spec_coef;
	int col_red, col_green, col_blue;

	int col_z;

	win_offset = xmax/2;


	local_scale = gl_win_size/xmax;

	y1 = (yy - win_offset)/SCALE * local_scale;
    x1 = (xx - win_offset)/SCALE * local_scale;

    //printf("x1 = %f, y1 = %f ", x1, y1);

    cuda_get_z_intersect_poly(x1, y1, &z, &n_z, disc_zero);


/*
    z_buf[x][y] = z;

    if (z > M_INF) {
    	n_buf[x][y] = n_z;
    } else {
    	n_buf[x][y] = 0;
    }
*/


	n = (z > M_INF) ? n_z : 0;

	if (z > M_INF) {
		color_red = 0.0;
		color_green = 0.0;
		color_blue = 0.0;

		if (n < 0) {
			color_red = -n/2;
			color_green = 0;
			color_blue = 0.0;
		} else if (n >= 0) {
			color_green = n/2;
			color_red = 0;
			color_blue = 0.0;
		}

		phong_kernel = 2 * n * n - 1;
		spec_coef = 0.3 * pow(phong_kernel, phong_exponent);

		color_red += spec_coef;
		color_green += spec_coef ;
		color_blue += spec_coef;

		col_red = (int) (250 * color_red);
		col_green = (int) (250 * color_green);
		col_blue = (int) (250 * color_blue);

#if 0

#define MAX_COL_Z (1 << 20)
#define NUM_STRIPES 32
#define STRIPE_PART 8

		col_z = (int)((z/15 + 1.0) * MAX_COL_Z);
		col_z %= MAX_COL_Z/NUM_STRIPES;
		if (0 <= col_z  && col_z <= MAX_COL_Z/(NUM_STRIPES * STRIPE_PART)) {
			col_blue = ::max(col_red, col_green);
			col_red = 0;
			col_green = 0;
		};

#endif

		mref(colmat_r_d, xx, yy) = col_red;
		mref(colmat_g_d, xx, yy) = col_green;
		mref(colmat_b_d, xx, yy) = col_blue;
	} else {
		// background color
		mref(colmat_r_d, xx, yy) = 250; //64;
		mref(colmat_g_d, xx, yy) = 250; //32;
		mref(colmat_b_d, xx, yy) = 250; //64;

	}

	__syncthreads();

}

#define THREAD_NUMXY 16

void gpu_compute_colmat(double a, double b, int xmax, int ymax, const cudaPoly3 & f3_h,
		double euler_phi, double euler_theta, double euler_psi,
		int *colmat_r, int *colmat_g, int *colmat_b) {

	int *colmat_r_d;
	int *colmat_g_d;
	int *colmat_b_d;
	
	int akt_deg_global_host = -1;
	
	for(int i = 0; i < 40; ++i) {
		int degi = f3_h.xexpo[i] + f3_h.yexpo[i] + f3_h.zexpo[i];
		if (degi > akt_deg_global_host) {
			akt_deg_global_host = degi;
		}
	}

	const int N = 4 * gl_win_size * gl_win_size;

	cout << "N = " << N << endl;

	cout << "xmax = " << xmax << " ymax = " << ymax << endl;

	//print_cudaPoly3(f3_h);

	printf("sizeof(cudaPoly3) = %ld\n ", sizeof(cudaPoly3));

	x_prepare_binom<<<1,1>>>();

	hipDeviceSynchronize();

	//x_print_binom<<<1,1>>>();

	CCE(hipDeviceSetLimit(hipLimitStackSize , 128 * 1024));

	CCE( hipMemcpyToSymbol(HIP_SYMBOL(akt_deg_global), &akt_deg_global_host, sizeof(int),
					0, hipMemcpyHostToDevice ) );

	CCE( hipMemcpyToSymbol(HIP_SYMBOL(f3), &f3_h, sizeof(cudaPoly3), 0, hipMemcpyHostToDevice) );

	hipDeviceSynchronize();


    cuda_rotate_mat<<<1,1>>>(euler_phi, euler_theta, euler_psi);

	cuda_init_f3_diff<<<1,1>>>();

	hipDeviceSynchronize();


	CCE( hipMalloc((void**) &colmat_r_d, N * sizeof(int) ) );
	CCE( hipMalloc((void**) &colmat_g_d, N * sizeof(int) ) );
	CCE( hipMalloc((void**) &colmat_b_d, N * sizeof(int) ) );

	dim3 grids(gl_win_size/THREAD_NUMXY, gl_win_size/THREAD_NUMXY);
	dim3 threads(THREAD_NUMXY, THREAD_NUMXY);

	compute_colmat<<<grids, threads>>>(a, b, xmax, ymax, euler_phi, euler_theta, euler_psi,
			colmat_r_d, colmat_g_d, colmat_b_d);

	cout << "computation done." << endl;


	CCE(hipMemcpy(colmat_r, colmat_r_d, N * sizeof(int), hipMemcpyDeviceToHost));
	CCE(hipMemcpy(colmat_g, colmat_g_d, N * sizeof(int), hipMemcpyDeviceToHost));
	CCE(hipMemcpy(colmat_b, colmat_b_d, N * sizeof(int), hipMemcpyDeviceToHost));

	CCE(hipFree(colmat_r_d));
	CCE(hipFree(colmat_g_d));
	CCE(hipFree(colmat_b_d));


}

